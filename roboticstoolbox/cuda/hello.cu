#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000


__device__ void add(float*out, float* a, float*b, int i);
__global__ void vector_add(float *out, float *a, float *b, int n);

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

__device__ void add(float*out, float* a, float*b, int i) {
    float *d_T;
    hipMalloc((void**)&d_T, sizeof(float));
    d_T[0] = a[i] + b[i];    
    out[i] = d_T[0];
    hipFree(d_T);
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n){
        add(out, a, b, tid);
        // out[tid] = a[tid] + b[tid];
    }
}


int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }


    // Allocate device memory 
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);


    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);

}

