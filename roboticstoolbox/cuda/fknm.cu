#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__device__ int _inv(double *m, double *invOut);
__device__ void mult(double *A, double *B, double *C);
__device__ void copy(double *A, double *B);
__device__ void _eye(double *data);




/* 
 * Params
 *  T: (N, 4, 4) the final transform matrix of all points (shared)
 *  tool: (N, 4, 4) the tool transform matrix of all points (shared)
 *  link_A: (cdim, 4, 4) the transformation matrix of all joints
 *  link_axes: (cdim, ): axes of all links
 *  link_isjoint: (cdim, ): 1/0 whether links are joints
 *  N: (int) number of points
 *  cdim: (int) number of joints
 *  out: (N, 6, cdim)
 */
__global__ void _jacob0(double *T,
                        double *tool, 
                        double *e_tool, 
                        double *link_A, 
                        int *link_axes,
                        int *link_isjoint, 
                        int N, 
                        int cdim, 
                        double *out)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    double *T_i, *tool_i;
    double *U, *temp, *etool_i;
    double *invU;
    double *link_iA;

    hipMalloc((void**)&U, sizeof(double) * 16);
    hipMalloc((void**)&invU, sizeof(double) * 16);
    hipMalloc((void**)&temp, sizeof(double) * 16);
    int j = 0;

    T_i = &T[tid * 16];
    tool_i = &tool[tid * 16];
    _eye(U);
    for (int i = 0; i < cdim; i++) {

        if (link_isjoint[i] == 1) {
            link_iA = &link_A[i * 16];
            mult(U, link_iA, temp);
            copy(temp, U);

            if (i == cdim - 1) {
                mult(U, etool_i, temp);
                copy(temp, U);
                mult(U, tool_i, temp);
                copy(temp , U);   
            }

            _inv(U, invU);
            mult(invU, T_i, temp);

            double *out_tid = &out[tid + 16];

            if (link_axes[i] == 0) {
                out_tid[0 * tid + j] = U[0 * 4 + 2] * temp[1 * 4 + 3] - U[0 * 4 + 1] * temp[2 * 4 + 3];
                out_tid[1 * tid + j] = U[1 * 4 + 2] * temp[1 * 4 + 3] - U[1 * 4 + 1] * temp[2 * 4 + 3];
                out_tid[2 * tid + j] = U[2 * 4 + 2] * temp[1 * 4 + 3] - U[2 * 4 + 1] * temp[2 * 4 + 3];
                out_tid[3 * tid + j] = U[0 * 4 + 2];
                out_tid[4 * tid + j] = U[1 * 4 + 2];
                out_tid[5 * tid + j] = U[2 * 4 + 2];
            }
            else if (link_axes[i] == 1)
            {
                out_tid[0 * tid + j] = U[0 * 4 + 0] * temp[2 * 4 + 3] - U[0 * 4 + 2] * temp[0 * 4 + 3];
                out_tid[1 * tid + j] = U[1 * 4 + 0] * temp[2 * 4 + 3] - U[1 * 4 + 2] * temp[0 * 4 + 3];
                out_tid[2 * tid + j] = U[2 * 4 + 0] * temp[2 * 4 + 3] - U[2 * 4 + 2] * temp[0 * 4 + 3];
                out_tid[3 * tid + j] = U[0 * 4 + 1];
                out_tid[4 * tid + j] = U[1 * 4 + 1];
                out_tid[5 * tid + j] = U[2 * 4 + 1];
            }
            else if (link_axes[i] == 2)
            {
                out_tid[0 * tid + j] = U[0 * 4 + 1] * temp[0 * 4 + 3] - U[0 * 4 + 0] * temp[1 * 4 + 3];
                out_tid[1 * tid + j] = U[1 * 4 + 1] * temp[0 * 4 + 3] - U[1 * 4 + 0] * temp[1 * 4 + 3];
                out_tid[2 * tid + j] = U[2 * 4 + 1] * temp[0 * 4 + 3] - U[2 * 4 + 0] * temp[1 * 4 + 3];
                out_tid[3 * tid + j] = U[0 * 4 + 2];
                out_tid[4 * tid + j] = U[1 * 4 + 2];
                out_tid[5 * tid + j] = U[2 * 4 + 2];
            }
            else if (link_axes[i] == 3)
            {
                out_tid[0 * tid + j] = U[0 * 4 + 0];
                out_tid[1 * tid + j] = U[1 * 4 + 0];
                out_tid[2 * tid + j] = U[2 * 4 + 0];
                out_tid[3 * tid + j] = 0.0;
                out_tid[4 * tid + j] = 0.0;
                out_tid[5 * tid + j] = 0.0;
            }
            else if (link_axes[i] == 4)
            {
                out_tid[0 * tid + j] = U[0 * 4 + 1];
                out_tid[1 * tid + j] = U[1 * 4 + 1];
                out_tid[2 * tid + j] = U[2 * 4 + 1];
                out_tid[3 * tid + j] = 0.0;
                out_tid[4 * tid + j] = 0.0;
                out_tid[5 * tid + j] = 0.0;
            }
            else if (link_axes[i] == 5)
            {
                out_tid[0 * tid + j] = U[0 * 4 + 2];
                out_tid[1 * tid + j] = U[1 * 4 + 2];
                out_tid[2 * tid + j] = U[2 * 4 + 2];
                out_tid[3 * tid + j] = 0.0;
                out_tid[4 * tid + j] = 0.0;
                out_tid[5 * tid + j] = 0.0;
            }
            j++;
        } else {
            link_iA = &link_A[i * 16];    
            mult(U, link_iA, temp);
            copy(temp, U);
        }
    }

    hipFree(U);
    hipFree(invU);
    hipFree(temp);
}


__device__ void _eye(double *data)
{
    data[0] = 1;
    data[1] = 0;
    data[2] = 0;
    data[3] = 0;
    data[4] = 0;
    data[5] = 1;
    data[6] = 0;
    data[7] = 0;
    data[8] = 0;
    data[9] = 0;
    data[10] = 1;
    data[11] = 0;
    data[12] = 0;
    data[13] = 0;
    data[14] = 0;
    data[15] = 1;
}

__device__ void copy(double *A, double *B)
{
    // copy A into B
    B[0] = A[0];
    B[1] = A[1];
    B[2] = A[2];
    B[3] = A[3];
    B[4] = A[4];
    B[5] = A[5];
    B[6] = A[6];
    B[7] = A[7];
    B[8] = A[8];
    B[9] = A[9];
    B[10] = A[10];
    B[11] = A[11];
    B[12] = A[12];
    B[13] = A[13];
    B[14] = A[14];
    B[15] = A[15];
}

__device__ void mult(double *A, double *B, double *C)
{
    const int N = 4;
    int i, j, k;
    double num;

    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            num = 0;
            for (k = 0; k < N; k++)
            {
                num += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = num;
        }
    }
}

__device__ int _inv(double *m, double *invOut)
{
    double *inv;
    hipMalloc((void**)&inv, sizeof(double) * 16);
    double det;
    int i;

    inv[0] = m[5] * m[10] * m[15] -
             m[5] * m[11] * m[14] -
             m[9] * m[6] * m[15] +
             m[9] * m[7] * m[14] +
             m[13] * m[6] * m[11] -
             m[13] * m[7] * m[10];

    inv[4] = -m[4] * m[10] * m[15] +
             m[4] * m[11] * m[14] +
             m[8] * m[6] * m[15] -
             m[8] * m[7] * m[14] -
             m[12] * m[6] * m[11] +
             m[12] * m[7] * m[10];

    inv[8] = m[4] * m[9] * m[15] -
             m[4] * m[11] * m[13] -
             m[8] * m[5] * m[15] +
             m[8] * m[7] * m[13] +
             m[12] * m[5] * m[11] -
             m[12] * m[7] * m[9];

    inv[12] = -m[4] * m[9] * m[14] +
              m[4] * m[10] * m[13] +
              m[8] * m[5] * m[14] -
              m[8] * m[6] * m[13] -
              m[12] * m[5] * m[10] +
              m[12] * m[6] * m[9];

    inv[1] = -m[1] * m[10] * m[15] +
             m[1] * m[11] * m[14] +
             m[9] * m[2] * m[15] -
             m[9] * m[3] * m[14] -
             m[13] * m[2] * m[11] +
             m[13] * m[3] * m[10];

    inv[5] = m[0] * m[10] * m[15] -
             m[0] * m[11] * m[14] -
             m[8] * m[2] * m[15] +
             m[8] * m[3] * m[14] +
             m[12] * m[2] * m[11] -
             m[12] * m[3] * m[10];

    inv[9] = -m[0] * m[9] * m[15] +
             m[0] * m[11] * m[13] +
             m[8] * m[1] * m[15] -
             m[8] * m[3] * m[13] -
             m[12] * m[1] * m[11] +
             m[12] * m[3] * m[9];

    inv[13] = m[0] * m[9] * m[14] -
              m[0] * m[10] * m[13] -
              m[8] * m[1] * m[14] +
              m[8] * m[2] * m[13] +
              m[12] * m[1] * m[10] -
              m[12] * m[2] * m[9];

    inv[2] = m[1] * m[6] * m[15] -
             m[1] * m[7] * m[14] -
             m[5] * m[2] * m[15] +
             m[5] * m[3] * m[14] +
             m[13] * m[2] * m[7] -
             m[13] * m[3] * m[6];

    inv[6] = -m[0] * m[6] * m[15] +
             m[0] * m[7] * m[14] +
             m[4] * m[2] * m[15] -
             m[4] * m[3] * m[14] -
             m[12] * m[2] * m[7] +
             m[12] * m[3] * m[6];

    inv[10] = m[0] * m[5] * m[15] -
              m[0] * m[7] * m[13] -
              m[4] * m[1] * m[15] +
              m[4] * m[3] * m[13] +
              m[12] * m[1] * m[7] -
              m[12] * m[3] * m[5];

    inv[14] = -m[0] * m[5] * m[14] +
              m[0] * m[6] * m[13] +
              m[4] * m[1] * m[14] -
              m[4] * m[2] * m[13] -
              m[12] * m[1] * m[6] +
              m[12] * m[2] * m[5];

    inv[3] = -m[1] * m[6] * m[11] +
             m[1] * m[7] * m[10] +
             m[5] * m[2] * m[11] -
             m[5] * m[3] * m[10] -
             m[9] * m[2] * m[7] +
             m[9] * m[3] * m[6];

    inv[7] = m[0] * m[6] * m[11] -
             m[0] * m[7] * m[10] -
             m[4] * m[2] * m[11] +
             m[4] * m[3] * m[10] +
             m[8] * m[2] * m[7] -
             m[8] * m[3] * m[6];

    inv[11] = -m[0] * m[5] * m[11] +
              m[0] * m[7] * m[9] +
              m[4] * m[1] * m[11] -
              m[4] * m[3] * m[9] -
              m[8] * m[1] * m[7] +
              m[8] * m[3] * m[5];

    inv[15] = m[0] * m[5] * m[10] -
              m[0] * m[6] * m[9] -
              m[4] * m[1] * m[10] +
              m[4] * m[2] * m[9] +
              m[8] * m[1] * m[6] -
              m[8] * m[2] * m[5];

    det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

    if (det == 0)
        return 0;

    det = 1.0 / det;

    for (i = 0; i < 16; i++)
        invOut[i] = inv[i] * det;

    hipFree(inv);
    return 1;
}



extern "C"{

/* 
 * Params
 *  T: (N, 4, 4) the final transform matrix of all points (shared)
 *  tool: (N, 4, 4) the end transform matrix of all points (shared)
 *  link_A: (cdim, 4, 4) the transformation matrix of all joints
 *  link_axes: (cdim, ): axes of all links
 *  link_isjoint: (cdim, ): 1/0 whether links are joints
 *  N: (int) number of points
 *  cdim: (int) number of joints
 *  out: (N, 6, cdim)
 */
void jacob0(double *T, 
            double *tool,
            double *etool,
            double *link_A, 
            int *link_axes,
            int *link_isjoint, 
            int N, 
            int cdim, 
            double *out)
    // affine_T[N]
    // link_axes[cdim]
    // link_A[cdim]
    // link_isjoint[cdim]
    // out
{
    double *d_T, *d_tool, *d_etool, *d_link_A;
    int *d_link_axes, *d_link_isjoint;
    double *d_out;

    hipMalloc((void**)&d_T, sizeof(double) * N * 16);
    hipMalloc((void**)&d_tool, sizeof(double) * N * 16);
    hipMalloc((void**)&d_etool, sizeof(double) * N * 16);
    hipMalloc((void**)&d_link_A, sizeof(double) * cdim * 16);
    hipMalloc((void**)&d_link_axes, sizeof(int) * cdim);
    hipMalloc((void**)&d_link_isjoint, sizeof(int) * cdim);
    hipMalloc((void**)&d_out, sizeof(double) * 6 * cdim);


    // Transfer data from host to device memory
    hipMemcpy(d_T, T, sizeof(double) * N * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_tool, tool, sizeof(double) * N * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_etool, etool, sizeof(double) * N * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_link_A, link_A, sizeof(double) * cdim * 16, hipMemcpyHostToDevice);
    hipMemcpy(d_link_axes, link_axes, sizeof(int) * cdim, hipMemcpyHostToDevice);
    hipMemcpy(d_link_isjoint, link_isjoint, sizeof(int) * cdim, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, sizeof(double) * 6 * cdim, hipMemcpyHostToDevice);


    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    _jacob0<<<grid_size,block_size>>>(d_T, 
                                      d_tool,
                                      d_etool,
                                      d_link_A, 
                                      d_link_axes,
                                      d_link_isjoint,
                                      N,
                                      cdim,
                                      d_out);

    // memset(out, 1, N * 6 * cdim);
    // out[0] = 1;
    hipMemcpy(out, d_out, sizeof(double) * 6 * cdim, hipMemcpyDeviceToHost);
    printf("Out size %d %d %f %f %f %f %f", N, cdim, out[0], out[1], out[2], out[3], out[4]);

    // Deallocate device memory
    hipFree(d_T);
    hipFree(d_tool);
    hipFree(d_etool);
    hipFree(d_link_A);
    hipFree(d_link_axes);
    hipFree(d_link_isjoint);
    hipFree(d_out);
}


}//extern "C"